﻿/*
============================================================================
Mantas Miežinas, IFF-2
LD4b
============================================================================
*/

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <fstream>
#include <iomanip>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

using namespace std;

//Konstantos
const char DataFile[] = "MiezinasM_L4.txt";
const int MAX_THREADS = 5;
const int MAX_ARRAY_SIZE = 5;

//Struktura gamintojo modeliams saugoti
struct model {
	char name[75];
	int quantity;
	double price;
};

//Konteinerine struktura saugoti modeliams
struct manufacturer {
	char name[15];
	int quantity;
	model models[MAX_ARRAY_SIZE];
};

//Funkciju prototipai
void ReadFile(string filename, thrust::host_vector<manufacturer> &AllModels);
void PrintTable(thrust::host_vector<manufacturer> printOut);
void PrintResults(thrust::host_vector<model> printOut);
void Plus(thrust::device_vector<manufacturer> &manu, thrust::device_vector<model> &resultsArray);


int main() {
	thrust::host_vector<manufacturer> AllModels(MAX_THREADS);
	thrust::host_vector<model> results(MAX_ARRAY_SIZE);

	ReadFile(DataFile, AllModels);
	PrintTable(AllModels);
	//Nusinuliname rezultatu masyvo elementus
	for (int i = 0; i < MAX_ARRAY_SIZE; i++){
		model data;
		strcpy(data.name, "");
		data.price = 0.0;
		data.quantity = 0;
		results[i] = data;
	}

	//Perkeliame duomenis is RAM i VRAM
	thrust::device_vector<manufacturer> manu = AllModels;
	thrust::device_vector<model> resultsArray = results;

	//Iskvieciame sumuojancia funkcija
	Plus(manu, resultsArray);
	hipDeviceSynchronize();

	//Persikeliame rezultatus is VRAM(GPU) i RAM(CPU)
	results = resultsArray;

	//Atspausdiname rezultatus
	PrintResults(results);
	system("Pause");
	return 0;
}

/*
============================================================================
ReadFile

Pradiniu duomenu nuskaitymo funkcija, per nuoroda grazina manufacturer
strukturos host vektoriu (CPU atmintyje)
============================================================================
*/
void ReadFile(string filename, thrust::host_vector<manufacturer> &AllModels) {
	string title;
	int count, j;
	j = 0;
	ifstream fin(filename);
	if (!fin) {
		cerr << "Couldn't open file!\n";
	}
	else {
		while (!fin.eof()){
			fin >> title >> count;
			strcpy(AllModels[j].name, title.c_str());
			//AllModels[j].name = title;
			AllModels[j].quantity = count;
			model models[MAX_ARRAY_SIZE];
			for (int i = 0; i < count; i++){
				model modelis;
				fin >> modelis.name >> modelis.quantity >> modelis.price;
				AllModels[j].models[i] = modelis;
			}
			j++;
		}
		fin.close();
	}
}

/*
============================================================================
PrintTable

Atspausdina pradinius duomenis lentelemis
============================================================================
*/
void PrintTable(thrust::host_vector<manufacturer> printOut){
	cout << "-----------------------------------------------------------------------------\n";
	for (manufacturer & manu : printOut){
		cout << right << setw(35) << manu.name << "\n";
		cout << "-----------------------------------------------------------------------------\n";
		cout << left << setw(63) << "Modelio Pavadinimas"
			<< setw(8) << "Kiekis"
			<< setw(5) << "Kaina" << "\n";
		cout << "-----------------------------------------------------------------------------\n";
		for (int i = 0; i < manu.quantity; i++){
			model forPrinting = manu.models[i];
			cout << left << setw(3) << to_string(i + 1) + ")"
				<< setw(60) << forPrinting.name
				<< setw(8) << forPrinting.quantity
				<< setw(5) << setprecision(4) << forPrinting.price
				<< "\n";
		}
		cout << "-----------------------------------------------------------------------------\n";
	}
}

/*
============================================================================
PrintResults

Atspausdina rezultatu vektoriu lenteleje
============================================================================
*/
void PrintResults(thrust::host_vector<model> printOut) {
	cout << "-----------------------------------------------------------------------------\n";
	cout << left << setw(63) << "Modelio Pavadinimas"
		<< setw(8) << "Kiekis"
		<< setw(5) << "Kaina" << "\n";
	cout << "-----------------------------------------------------------------------------\n";
	for (int i = 0; i < MAX_ARRAY_SIZE; i++) {
		model forPrinting = printOut[i];
		cout << left << setw(3) << to_string(i + 1) + ")"
			<< setw(60) << forPrinting.name
			<< setw(8) << forPrinting.quantity
			<< setw(5) << setprecision(4) << forPrinting.price
			<< "\n";
	}
}

/*
============================================================================
Plus

Susumuoja pradiniu duomenu masyvus i viena masyva ir grazina rezultata per
nuoroda. Naudojami device vektoriai, reiskias naudojama atmintis priklauso
GPU.
============================================================================
*/
void Plus(thrust::device_vector<manufacturer> &manu, thrust::device_vector<model> &resultsArray) {
	for (int i = 0; i < manu.size(); i++) {
		for (int j = 0; j < resultsArray.size(); j++) {
			model data = resultsArray[i];
			manufacturer addData = manu[j];
			strcat(data.name, addData.models[i].name);
			data.price += addData.models[i].price;
			data.quantity += addData.models[i].quantity;
			resultsArray[i] = data;
		}
	}
}