#include "hip/hip_runtime.h"
﻿/*
============================================================================
Mantas Miežinas, IFF-2
LD4a
============================================================================
*/

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <fstream>
#include <iomanip>

using namespace std;

//Konstantos
const char DataFile[] = "MiezinasM_L4.txt";
const int MAX_THREADS = 5;
const int MAX_ARRAY_SIZE = 5;

//Struktura gamintojo modeliams saugoti
struct model {
	char name[75];
	int quantity;
	double price;
};

//Konteinerine struktura saugoti modeliams
struct manufacturer {
	char name[15];
	int quantity;
	model models[MAX_ARRAY_SIZE];
};

//Funkciju prototipai
void ReadFile(string filename, manufacturer (&AllModels)[MAX_THREADS]);
void PrintTable(manufacturer(&printOut)[MAX_THREADS]);
void PrintResults(model (&printOut)[MAX_ARRAY_SIZE]);

__global__ void RunOnGPU(manufacturer *printOut, model *resultsArray);
__device__ char * my_strcpy(char *dest, const char *src);
__device__ char * my_strcat(char *dest, const char *src);

int main() {
	manufacturer AllModels[MAX_THREADS];
	model results[MAX_ARRAY_SIZE];

	ReadFile(DataFile, AllModels);
	PrintTable(AllModels);
	//Nusinuliname rezultatu masyvo elementus
	for (int i = 0; i < MAX_ARRAY_SIZE; i++){
		model data;
		strcpy(data.name, "");
		data.price = 0.0;
		data.quantity = 0;
		results[i] = data;
	}

	//Paruosiame GPU, t.y. perkeliame duomenis is RAM i VRAM
	manufacturer *manu;
	model *resultsArray;
	
	int i = MAX_THREADS * sizeof(manufacturer);
	int j = MAX_ARRAY_SIZE * sizeof(model);

	hipMalloc((void**)&resultsArray , j);
	hipMalloc((void**)&manu, i);

	hipMemcpy(resultsArray, results, j, hipMemcpyHostToDevice);
	hipMemcpy(manu, AllModels, i, hipMemcpyHostToDevice);

	//Iskvieciame GPU metoda, sumuojanti masyvu elementus
	RunOnGPU << <1, MAX_THREADS >> >(manu, resultsArray);
	hipDeviceSynchronize();

	//Perkeliame rezultatus is VRAM (GPU) i RAM (CPU) atminti
	hipMemcpy(results, resultsArray, j, hipMemcpyDeviceToHost);

	//Atlaisviname VRAM atminti
	hipFree(manu);
	hipFree(resultsArray);

	//Atspausdiname rezultatus
	PrintResults(results);
	system("Pause");
	return 0;
}

/*
============================================================================
ReadFile

Pradiniu duomenu nuskaitymo funkcija, per nuoroda grazina manufacturer
strukturos masyva
============================================================================
*/
void ReadFile(string filename, manufacturer (&AllModels)[MAX_THREADS]) {
	string title;
	int count, j;
	j = 0;
	ifstream fin(filename);
	if (!fin) {
		cerr << "Couldn't open file!\n";
	}
	else {
		while (!fin.eof()){
			fin >> title >> count;
			strcpy(AllModels[j].name, title.c_str());
			//AllModels[j].name = title;
			AllModels[j].quantity = count;
			model models[MAX_ARRAY_SIZE];
			for (int i = 0; i < count; i++){
				model modelis;
				fin >> modelis.name >> modelis.quantity >> modelis.price;
				AllModels[j].models[i] = modelis;
			}
			j++;
		}
		fin.close();
	}
}

/*
============================================================================
PrintTable

Atspausdina pradinius duomenis lentelemis
============================================================================
*/
void PrintTable(manufacturer(&printOut)[MAX_THREADS]){
	cout << "-----------------------------------------------------------------------------\n";
	for (manufacturer & manu : printOut){
		cout << right << setw(35) << manu.name << "\n";
		cout << "-----------------------------------------------------------------------------\n";
		cout << left << setw(63) << "Modelio Pavadinimas"
			<< setw(8) << "Kiekis"
			<< setw(5) << "Kaina" << "\n";
		cout << "-----------------------------------------------------------------------------\n";
		for (int i = 0; i < manu.quantity; i++){
			model forPrinting = manu.models[i];
			cout << left << setw(3) << to_string(i + 1) + ")"
				<< setw(60) << forPrinting.name
				<< setw(8) << forPrinting.quantity
				<< setw(5) << setprecision(4) << forPrinting.price
				<< "\n";
		}
		cout << "-----------------------------------------------------------------------------\n";
	}
}

/*
============================================================================
PrintResults

Atspausdina rezultatu masyva lenteleje
============================================================================
*/
void PrintResults(model(&printOut)[MAX_ARRAY_SIZE]) {
	cout << "-----------------------------------------------------------------------------\n";
	cout << left << setw(63) << "Modelio Pavadinimas"
		<< setw(8) << "Kiekis"
		<< setw(5) << "Kaina" << "\n";
	cout << "-----------------------------------------------------------------------------\n";
	for (int i = 0; i < MAX_ARRAY_SIZE; i++) {
		model forPrinting = printOut[i];
		cout << left << setw(3) << to_string(i + 1) + ")"
			<< setw(60) << forPrinting.name
			<< setw(8) << forPrinting.quantity
			<< setw(5) << setprecision(4) << forPrinting.price
			<< "\n";
	}
}

/*
============================================================================
RunOnGPU

Susumuoja pradiniu duomenu masyvus i viena masyva. Rezultatas lieka GPU
atmintyje, is kur veliau ji paimsime.
============================================================================
*/
__global__ void RunOnGPU(manufacturer *printOut, model *resultsArray) {
	int gija = threadIdx.x;
	for (int i = 0; i < MAX_THREADS; i++) {
		my_strcat(resultsArray[gija].name, printOut[i].models[gija].name);
		resultsArray[gija].price += printOut[i].models[gija].price;
		resultsArray[gija].quantity += printOut[i].models[gija].quantity;
	}
}

/*
============================================================================
Pagalbines funkcijos, my_strcat skirta apjungti dviems simboliu eilutems,
ji naudoja my_strcpy. Ju reikia tam, nes CUDA nepalaiko string.h
bibliotekos, kurioje yra strcat() funkcija.
============================================================================
*/
__device__ char * my_strcpy(char *dest, const char *src){
	int i = 0;
	do {
		dest[i] = src[i];
	} while (src[i++] != 0);
	return dest;
}

__device__ char * my_strcat(char *dest, const char *src){
	int i = 0;
	while (dest[i] != 0) i++;
	my_strcpy(dest + i, src);
	return dest;
}