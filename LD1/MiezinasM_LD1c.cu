#include "hip/hip_runtime.h"
/**
* 1. Kiek iteracij� i� eil�s padaro vienas procesas?
* Ats.: vien� pilnai
*
* 2. Kokia tvarka atspausdinami to paties masyvo duomenys?
* Ats.: tokia, kokia sura�yti
*
* 3. Kokia tvarka vykdomi procesai?
* Ats.: tokia, kokia startuoja(tai yra 1 procesas atlieka viena iteracija, tuomet 2 procesas ir t.t.,
* kol atliktos visos iteracijos
*/

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <fstream>
#include <iomanip>

using namespace std;

//Konstantos
const char DataFile[] = "MiezinasM.txt";
const int MAX_THREADS = 5;
const int MAX_ARRAY_SIZE = 10;

//Struktura gamintojo modeliams saugoti
struct model {
	char name[15];
	int quantity;
	double price;
};

//Gamintojo klase, tarnauja kaip modeliu konteineris
class Manufacturer {
public:
	string name;
	int quantity;
	model models[MAX_ARRAY_SIZE];
	//Konstruktorius
	Manufacturer(string name, int quantity, model models[]) {
		this->name = name;
		this->quantity = quantity;
		for (int i = 0; i < quantity; i++){
			this->models[i] = models[i];
		}
	}
	Manufacturer() {
		this->name = "";
		this->quantity = 0;
	}
};

//Funkciju prototipai
void ReadFile(string filename, Manufacturer(&AllModels)[MAX_THREADS]);
void PrintTable(Manufacturer(&printOut)[MAX_THREADS]);
__device__ void PrintManufacturerModels(Manufacturer printOut, int procNum);
__global__ void RunOnGPU(Manufacturer *printOut);

int main() {
	Manufacturer AllModels[MAX_THREADS];
	ReadFile(DataFile, AllModels);
	PrintTable(AllModels);
	//Paruosiame GPU, t.y. perkeliame duomenis is RAM i VRAM
	Manufacturer *manu;
	int i = MAX_THREADS * sizeof(Manufacturer);
	hipMalloc((void**)&manu, i);
	hipMemcpy(manu, AllModels, i, hipMemcpyHostToDevice);
	//Iskvieciame GPU
	RunOnGPU << <1, MAX_THREADS >> >(manu);
	hipDeviceSynchronize();
	//Atlaisviname VRAM atminti
	hipFree(manu);

	system("Pause");
	return 0;
}

void ReadFile(string filename, Manufacturer(&AllModels)[MAX_THREADS]) {
	string title;
	int count, j;
	j = 0;
	ifstream fin(filename);
	if (!fin) {
		cerr << "Couldn't open file!\n";
	}
	else {
		while (!fin.eof()){
			fin >> title >> count;
			AllModels[j].name = title;
			AllModels[j].quantity = count;
			model models[MAX_ARRAY_SIZE];
			for (int i = 0; i < count; i++){
				model modelis;
				fin >> modelis.name >> modelis.quantity >> modelis.price;
				AllModels[j].models[i] = modelis;
			}
			j++;
		}
		fin.close();
	}
}
void PrintTable(Manufacturer(&printOut)[MAX_THREADS]){
	cout << "-----------------------------------------------------------------------------\n";
	for (Manufacturer & manu : printOut){
		cout << right << setw(35) << manu.name << "\n";
		cout << "-----------------------------------------------------------------------------\n";
		cout << left << setw(63) << "Modelio Pavadinimas"
			<< setw(8) << "Kiekis"
			<< setw(5) << "Kaina" << "\n";
		cout << "-----------------------------------------------------------------------------\n";
		for (int i = 0; i < manu.quantity; i++){
			model forPrinting = manu.models[i];
			cout << left << setw(3) << to_string(i + 1) + ")"
				<< setw(60) << forPrinting.name
				<< setw(8) << forPrinting.quantity
				<< setw(5) << setprecision(4) << forPrinting.price
				<< "\n";
		}
		cout << "-----------------------------------------------------------------------------\n";
	}
}
__device__ void PrintManufacturerModels(Manufacturer printOut, int procNum){
	for (int i = 0; i < printOut.quantity; i++) {
		model mod = printOut.models[i];
		printf("procesas_%d %d    %15s %2d %4.2f\n", procNum + 1, i + 1, mod.name, mod.quantity, mod.price);
	}
}
__global__ void RunOnGPU(Manufacturer *printOut) {
	PrintManufacturerModels(printOut[threadIdx.x], threadIdx.x);
}